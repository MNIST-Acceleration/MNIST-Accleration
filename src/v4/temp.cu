#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>;

#define CHECK_CUDA(call)                                                    
{                                                                       
    hipError_t err = call;                                             
    if (err != hipSuccess) {                                           
        std::cerr << "CUDA error: " << hipGetErrorString(err) << "\n";
        exit(EXIT_FAILURE);                                             
    }                                                                   
}

#define CHECK_CUBLAS(call)                                                  
{                                                                       
    hipblasStatus_t stat = call;                                         
    if (stat != HIPBLAS_STATUS_SUCCESS) {                                
        std::cerr << "cuBLAS error\n";                                  
        exit(EXIT_FAILURE);                                             
    }                                                                   
}

int main() {
    const int N = 128;  // Hidden layer size
    const int input_size = 784;  // Input size (28x28 flattened)

    // Host matrices in half precision (FP16)
    half input[input_size];      // 1 x 784
    half weights[input_size * N];  // 784 x 128
    half bias[N];                // 1 x 128
    half output[N];              // 1 x 128

    // Initialize matrices with ones
    for (int i = 0; i < input_size; ++i) {
        input[i] = __float2half(1.0f);  // All ones
    }

    for (int i = 0; i < input_size * N; ++i) {
        weights[i] = __float2half(1.0f);  // All ones
    }

    for (int i = 0; i < N; ++i) {
        bias[i] = __float2half(1.0f);  // All ones
    }

    // Device pointers
    half *d_input, *d_weights, *d_bias, *d_output;

    CHECK_CUDA(hipMalloc(&d_input, sizeof(half) * input_size));
    CHECK_CUDA(hipMalloc(&d_weights, sizeof(half) * input_size * N));
    CHECK_CUDA(hipMalloc(&d_bias, sizeof(half) * N));
    CHECK_CUDA(hipMalloc(&d_output, sizeof(half) * N));

    CHECK_CUDA(hipMemcpy(d_input, input, sizeof(half) * input_size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_weights, weights, sizeof(half) * input_size * N, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_bias, bias, sizeof(half) * N, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Enable Tensor Core operations
    CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    float alpha = 1.0f;
    float beta = 0.0f;

    // Matrix multiplication: output = input * weights + bias
    // Using hipblasGemmEx: C = alpha * A * B + beta * C
    CHECK_CUBLAS(hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,  // No transpose on A and B
        N, 1, input_size,          // M = N, N = 1 (batch size), K = input_size (784)
        &alpha,
        d_weights, HIP_R_16F, input_size,  // Weights (784 x 128), leading dimension = input_size
        d_input, HIP_R_16F, input_size,   // Input (1 x 784), leading dimension = input_size
        &beta,
        d_output, HIP_R_16F, N,  // Output (1 x 128), leading dimension = N (hidden size)
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    ));

    // Add the bias to the output (output += bias)
    // Bias is a vector and needs to be added to each element in the output vector
    float alpha_bias = 1.0f;
    CHECK_CUBLAS(hipblasSaxpy(handle, N, &alpha_bias, d_bias, 1, d_output, 1));

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(output, d_output, sizeof(half) * N, hipMemcpyDeviceToHost));

    // Print result
    std::cout << "Output of the first hidden layer (using Tensor Cores):\n";
    for (int i = 0; i < N; ++i) {
        std::cout << __half2float(output[i]) << " ";
        if ((i + 1) % 16 == 0) std::cout << std::endl;
    }

    // Cleanup
    hipFree(d_input); 
    hipFree(d_weights); 
    hipFree(d_bias); 
    hipFree(d_output); 
    hipblasDestroy(handle);

    return 0;
}
